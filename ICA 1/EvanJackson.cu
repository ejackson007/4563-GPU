#include "hip/hip_runtime.h"
//Evan Jackson
#include <stdio.h>

#define SIZE 10240

__global__
void vecProd(int* aDevice, int* bDevice, int* cDevice){
    int i = threadIdx.x + blockDim.x * blockIdx.x; // get location of thread
    if(i < SIZE){
        cDevice[i] = aDevice[i] * bDevice[i];//add to c and save to total
    }
}

//host code
int main(){
    //create variable to create arrays
    size_t arraySize = SIZE * sizeof(int);
    int total = 0;
    //host vector
    int* aHost; 
    int* bHost;
    int* cHost;
    //device vector
    int* aDevice;
    int* bDevice;
    int* cDevice;

    //allocate for host
    aHost = (int*)malloc(arraySize);
    bHost = (int*)malloc(arraySize);
    cHost = (int*)malloc(arraySize);
    //fill array aHost and bHost
    for(int i = 0; i < SIZE; i++){
        aHost[i] = 2*i;
        Host[i] = 2*1 + 1;
    }

    //allocate memory for device and transfer to device
    hipMalloc(&aDevice, arraySize);
    hipMalloc(&bDevice, arraySize);
    hipMalloc(&cDevice, arraySize);
    hipMemcpy(aDevice, aHost, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, bHost, arraySize, hipMemcpyHostToDevice);

    //maximum amount of threads, with 2 blocks.
    dim3 dimGrid(2,1,1);
    dim3 dimBlock(1024,1,1);

    //call gpu process
    vecProd<<<dimGrid,dimBlock>>>(aDevice, bDevice, cDevice);

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);

    //print for 2 block size
    printf("2 Blocks (first,last) = {%d, %d)\n", cHost[0], cHost[SIZE - 1]);
    //reinitiliaze a and b
    for(int i = 0; i < SIZE; i++){
        aHost[i] = 2*i;
        Host[i] = 2*1 + 1;
    }
    //reallocate device memory
    hipMemcpy(aDevice, aHost, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, bHost, arraySize, hipMemcpyHostToDevice);

    //repeat for 10 blocks

    //maximum amount of threads, with 10 blocks.
    dim3 dimGrid(10,1,1);
    dim3 dimBlock(1024,1,1);

    //call gpu process
    vecProd<<<dimGrid,dimBlock>>>(aDevice, bDevice, cDevice);

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);

    //free device memory;
    hipFree(aDevice);
    hipFree(bDevice);
    hipFree(cDevice);
    //print first and last for 10 blocks
    printf("10 Blocks (first,last) = {%d, %d)\n", cHost[0], cHost[SIZE - 1]);

    return 0;
}