//Evan Jackson

#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 10240

__global__
void vecProdCyclic(int* aDevice, int* bDevice, int* cDevice, int block){
    int i = threadIdx.x + blockDim.x * blockIdx.x; // get location of thread
    int jump = SIZE/block;//creates jump for each thread to make
    if(i < SIZE){
        for(int x = 1; x <= 5; x++)
            cDevice[i + (jump*x)] = aDevice[i + (jump*x)] * bDevice[i + (jump*x)];//add to c and save to total
    }
}

__global__
void vecProdNonCyclic(int* aDevice, int* bDevice, int* cDevice, int block){
    int i = threadIdx.x + blockDim.x * blockIdx.x; // get location of thread
    if(i < SIZE){
        cDevice[i + block] = aDevice[i + block] * bDevice[i + block];//add to c and save to total
    }
}

//host code
int main(){
    //create variable to create arrays
    size_t arraySize = SIZE * sizeof(int);
    //host vector
    int* aHost; 
    int* bHost;
    int* cHost;
    //device vector
    int* aDevice;
    int* bDevice;
    int* cDevice;

    //allocate for host
    aHost = (int*)malloc(arraySize);
    bHost = (int*)malloc(arraySize);
    cHost = (int*)malloc(arraySize);
    //fill array aHost and bHost
    for(int i = 0; i < SIZE; i++){
        aHost[i] = 2*i;
        bHost[i] = 2*i + 1;
    }

    //allocate memory for device and transfer to device
    hipMalloc(&aDevice, arraySize);
    hipMalloc(&bDevice, arraySize);
    hipMalloc(&cDevice, arraySize);
    hipMemcpy(aDevice, aHost, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, bHost, arraySize, hipMemcpyHostToDevice);

 /**********************************************************************************
 ******************** 2 Blocks, Non Cyclic *****************************************
 **********************************************************************************/  

    //maximum amount of threads, with 2 blocks.
    dim3 dimGrid2(2,1,1);
    dim3 dimBlock(1024,1,1);

    //call gpu process
    //Breaks up the full array into processable
    int jump = (SIZE / 1024) / 2;
    for(int i = 0; i < jump; i++){
        vecProdNonCyclic<<<dimGrid2,dimBlock>>>(aDevice, bDevice, cDevice, i*(SIZE/jump));
    }

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);

    //print for 2 block size
    printf("2 Blocks - Cyclic(C[0], C[10239]) = {%d, %d)\n", cHost[0], cHost[SIZE - 1]);

    //reset C array
    for(int i = 0; i < SIZE; i++){
        cHost[i] = 0;
    }
    hipMemcpy(cDevice, cHost, arraySize, hipMemcpyHostToDevice);


/**********************************************************************************
 ******************** 2 Blocks, Cyclic ********************************************
 **********************************************************************************/ 

    //call gpu process
    vecProdCyclic<<<dimGrid2,dimBlock>>>(aDevice, bDevice, cDevice, jump);

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);

    //print for 2 block size
    printf("2 Blocks - Cyclic(C[0], C[10239]) = {%d, %d)\n", cHost[0], cHost[SIZE - 1]);

    //reset C array
    for(int i = 0; i < SIZE; i++){
        cHost[i] = 0;
    }
    hipMemcpy(cDevice, cHost, arraySize, hipMemcpyHostToDevice);

/**********************************************************************************
 ******************** 10, Blocks ********************************************
 **********************************************************************************/ 

    //maximum amount of threads, with 10 blocks.
    dim3 dimGrid10(10,1,1);

    //call gpu process
    jump = (SIZE / 1024) / 10;
    for(int i = 0; i < jump; i++){
        vecProdNonCyclic<<<dimGrid10,dimBlock>>>(aDevice, bDevice, cDevice, i*(SIZE/jump));
    }

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);

    //free device memory;
    hipFree(aDevice);
    hipFree(bDevice);
    hipFree(cDevice);
    //print first and last for 10 blocks
    printf("10 Blocks - (C[0], C[10239]) = {%d, %d)\n", cHost[0], cHost[SIZE - 1]);

    return 0;
}