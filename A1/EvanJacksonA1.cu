//Evan Jackson

#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 4096

__global__
void vecAdd(int* aDevice, int* bDevice, int* cDevice){
    int i = threadIdx.x + blockDim.x * blockIdx.x; // get location of thread
    if(i < SIZE){
        cDevice[i] = aDevice[i] + bDevice[i];//add to c and save to total
    }
}

//host code
int main(){
    size_t arraySize = SIZE * sizeof(int);
    int total = 0;
    //host vector
    int* aHost; 
    int* bHost;
    int* cHost;
    //device vector
    int* aDevice;
    int* bDevice;
    int* cDevice;

    //allocate for host and store
    aHost = (int*)malloc(arraySize);
    bHost = (int*)malloc(arraySize);
    cHost = (int*)malloc(arraySize);
    //fill array aHost
    for(int i = 0; i < SIZE; i++){
        aHost[i] = i;
    }
    //fill array bHost
    for(int i = 0; i < SIZE; i++){
        bHost[i] = 4095 + i;
    }

    //allocate memory for device and transfer to device
    hipMalloc(&aDevice, arraySize);
    hipMalloc(&bDevice, arraySize);
    hipMalloc(&cDevice, arraySize);
    hipMemcpy(aDevice, aHost, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, bHost, arraySize, hipMemcpyHostToDevice);

    dim3 dimGrid(4,1,1); //1D of 4 block, so that each block will have maximum threads
    dim3 dimBlock(1024,1,1);

    //call gpu process
    vecAdd<<<dimGrid,dimBlock>>>(aDevice, bDevice, cDevice);

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);
    //free device memory;
    hipFree(aDevice);
    hipFree(bDevice);
    hipFree(cDevice);
    //find total of vector
    for(int i = 0; i < SIZE; i++){
        total += cHost[i];
    }

    //print values
    printf("First element of vector C: %i\n", cHost[0]);
    printf("Last element of vector C: %i\n", cHost[SIZE - 1]);
    printf("Summation of Elements in vector C: %i\n", total);

    return 0;
}