#include "hip/hip_runtime.h"
//Evan Jackson
#include <stdio.h>

#define SIZE 4096

float total = 0;

__global__
void vecAdd(int* aDevice, int* bDevice, int* cDevice){
    int i = threadIdx.x + blockDim.x * blockId.x; // get location of thread
    if(i < SIZE){
        cDevice[i] = aDevice[i] + bDevice[i];//add to c and save to total
        total += cDevice[i];
    }
}

//host code
int main(){
    int arraySize = SIZE * sizeof(int);
    int* aHost, bHost, cHost, aDevice, bDevice, cDevice;

    //allocate for host and store
    aHost = (int*)malloc(arraySize);
    bHost = (int*)malloc(arraySize);
    cHost = (int*)malloc(arraySize);
    //fill array aHost
    for(int i = 0; i < SIZE; i++){
        aHost[i] = i;
    }
    //fill array bHost
    for(int i = 0; i < SIZE; i++){
        bHost[i] = 4905 + i;
    }

    //allocate memory for device and transfer to device
    hipMalloc(&aDevice, arraySize);
    hipMalloc(&bDevice, arraySize);
    hipMalloc(&cDevice, arraySize)
    hipMemcpy(aDevice, aHost, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, bHost, arraySize, hipMemcpyHostToDevice);

    dim3 dimGrid(4,1,1); //1D of 4 block, so that each block will have maximum threads
    dim3 dimBlock(1024,1,1);

    //call gpu process
    vecAdd<<<dimGrid,dimBlock>>>(aDevice, bDevice, cDevice);

    //transfer back to host
    hipMemcpy(cHost, cDevice, arraySize, hipMemcpyDeviceToHost);
    //free device memory;
    hipFree(aDevice);
    hipFree(bDevice);
    hipFree(cDevice);
    //print values
    printf("First element of vector C: %i\n", cHost[0]);
    printf("Last element of vector C: %i\n", cHost[SIZE - 1]);
    printf("Summation of Elements in vector C: %i\n", total);

    return 0;
}