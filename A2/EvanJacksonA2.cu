//*****************************************************************************
// Assignment #2
// Evan Jackson and Joseph Williamson
// GPU Programming Date: (10/15/2020)
//*****************************************************************************
// This program solves a Fast Fourier Transform using the Cooley-Tukey
// Algorithm, also known as Radix-2. It does this by recursively cutting the 
// table in half creating an "Even" and "Odd" part until they are individuals
// in which case it goes back up the stack operating the function on each
// layer. The GPU is taken advantage of here by working on the arithmetic
// part of the algorithm, as it is the most time consuming part. Given
// the amount of threads, each layer arithmetic can be handled virtually 
// O(1) time. The input is hard coded in the program, and therefore needs 
// no input file. 
// 
// TO RUN PROGRAM:
//
// 1) Put EvanJacksonJosephWilliamson.cu and A2Script in the same directory
//    in maverick2
// 2) run command "sbatch A2Script" in terminal while in the same directory as
//    the files. 
// 3) results will be show in file named "mysimplejob.xxxxxx.out"
//****************************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>//used for cos, sin, and PI

#define SIZE 8192
//complex.h is not included in cuda, therefore our own
//complex struct is created.
typedef struct complex_t {
    double real;
    double imag;
} complex;

/*
__device__ complex dPolar()
Parameters: double theta
This is a function that can only be called by the device. This
function carries out the e^(-2.0*M_PI*i/n) part of the FFT function.
theta == (-2.0*M_PI*i/n) where i is the thread id
returns the results of the operation. 
*/
__device__ complex dPolar(double theta){
    complex result;
    result.real = cos(theta);
    result.imag = sin(theta);
    return result;
}

/*
__device__ complex dAdd()
*/
__device__ complex dAdd(complex l, complex r){
    complex result;
    result.real = l.real + r.real;
    result.imag = l.imag + r.imag;
    return result;
}


__device__ complex dSubtract(complex l, complex r){
    complex result;
    result.real = l.real - r.real;
    result.imag = l.imag - r.imag;
    return result;
}


__device__ complex dMultiply(complex l, complex r){
    complex result;
    result.real = l.real*r.real - l.imag*r.imag;
    result.imag = l.real*r.imag + l.imag*r.real;
    return result;
}

__global__
void solveOdd(complex * oddDevice, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n/2){
        oddDevice[i] = dMultiply(dPolar(-2.0*M_PI*i/n), oddDevice[i]);
    }
}

__global__
void solveX(complex * oddDevice, complex * evenDevice, complex * XDevice, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n/2){
        XDevice[i] = dAdd(evenDevice[i], oddDevice[i]);
        XDevice[i + n/2] = dSubtract(evenDevice[i], oddDevice[i]);
    }
}

complex *fillArray(){
    complex *all_nums = (complex *)malloc(sizeof(struct complex_t) * SIZE);
    all_nums[0].real = 3.6;
    all_nums[0].imag = 2.6;
    all_nums[1].real = 2.9;
    all_nums[1].imag = 6.3;
    all_nums[2].real = 5.6;
    all_nums[2].imag = 4.0;
    all_nums[3].real = 4.8;
    all_nums[3].imag = 9.1;
    all_nums[4].real = 3.3;
    all_nums[4].imag = 0.4;
    all_nums[5].real = 5.9;
    all_nums[5].imag = 4.8;
    all_nums[6].real = 5.0;
    all_nums[6].imag = 2.6;
    all_nums[7].real = 4.3;
    all_nums[7].imag = 4.1;
    for(int i = 8; i < SIZE; i++){
        all_nums[i].real = 0;
        all_nums[i].imag = 0;
    }
    return all_nums;
}

complex *CT_FFT(complex* table, int n){
    int arraySize = sizeof(struct complex_t) * n;
    complex *X = (complex *)malloc(sizeof(struct complex_t) * n);
    complex *odd, *even, *ODD, *EVEN, *XDevice, *oddDevice, *evenDevice;

    if(n == 1){
        X[0] = table[0];
        return X;
    }

    even = (complex *)malloc(sizeof(struct complex_t) * n/2);
    odd = (complex *)malloc(sizeof(struct complex_t) * n/2);
    for(int i = 0; i < n/2; i++){
        even[i] = table[2*i];
        odd[i] = table[2*i + 1];
    }

    EVEN = CT_FFT(even, n/2);
    ODD = CT_FFT(odd, n/2);

    //start the mess
    hipMalloc(&evenDevice, arraySize/2);
    hipMalloc(&oddDevice, arraySize/2);
    hipMalloc(&XDevice, arraySize);
    hipMemcpy(evenDevice, EVEN, arraySize/2, hipMemcpyHostToDevice);
    hipMemcpy(oddDevice, ODD, arraySize/2, hipMemcpyHostToDevice);

    dim3 dimGrid(4,1,1);
    dim3 dimBlock(1024,1,1);

    solveOdd<<<dimGrid, dimBlock>>>(oddDevice, n);
    hipDeviceSynchronize();
    solveX<<<dimGrid, dimBlock>>>(oddDevice, evenDevice, XDevice, n);
    hipDeviceSynchronize();

    hipMemcpy(X, XDevice, arraySize, hipMemcpyDeviceToHost);
    free(EVEN);
    free(ODD);
    hipFree(oddDevice);
    hipFree(evenDevice);
    hipFree(XDevice);
    return X;
}

int main(){
    complex *table, *printing;
    table = fillArray();
    printing = CT_FFT(table, SIZE);
    printf("TOTAL PROCESSED SAMPLES: %d\n", SIZE);
    printf("=====================================\n");
    for(int i=0; i < 8; i++){
        //print real and imaginary values
        printf("XR[%d]: %f  XI[%d]: %fi\n", i, printing[i].real, i, printing[i].imag);
        printf("=====================================\n");
    }
    return 0;
}
