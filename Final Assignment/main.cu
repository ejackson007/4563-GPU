// Calculate Mandelbrot in Cuda

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

//.0 used fo program uses it as double
#define WIDTH 1920.0
#define HEIGHT 1080.0
#define area WIDTH * HEIGHT
#define xa -2.0
#define xb 1.0
#define ya -1.0
#define yb 1.0
#define maxIt 255

typedef struct rgb{
    int r;
    int g;
    int b;
} pixel;
typedef struct complex_t {
    double real;
    double imag;
} complex;

__device__ double absComplex(complex z){
    return sqrt(z.real*z.real + z.imag*z.imag);
}

__device__ complex sqComplex(complex z){
    complex result;
    result.real = z.real*z.real - z.imag*z.imag;
    result.imag = z.real*z.imag + z.imag*z.real;
    return result;
}

__device__ complex addComplex(complex z, complex c){
    complex result;
    result.real = z.real + c.real;
    result.imag = z.imag + c.imag;
    return result;
}

__global__
void mandel(pixel * image){
    //get thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    double zy, zx;
    if(i < area){
        double x = i / HEIGHT;
        double y = i % int(HEIGHT); //mod of floating point
        int j;
        complex z, c;
        zy = y * (yb - ya) / (HEIGHT - 1) + ya;
        zx = x * (xb - xa) / (WIDTH - 1) + xa;
        z.real = zx;
        z.imag = zy;
        c = z;
        for(j = 0; j < maxIt; j++){
            if(absComplex(z) > 2){
                break;
            }
            z = addComplex(sqComplex(z), c);
        }
        image[i].r = j % 4 * 64;
        image[i].g = j % 8 * 32;
        image[i].b = j % 16 * 16;
    }
}

int main(){
    clock_t start, end;
    double cpu_time_used;
    long int imageSize = sizeof(struct rgb) * area;
    pixel *image = (pixel *)malloc(imageSize), *imageDevice;
    
    hipMalloc(&imageDevice, imageSize);

    dim3 gridDim(area/1024, 1, 1);
    dim3 dimBlock(1024,1,1);

    start = clock();
    mandel<<<gridDim, dimBlock>>>(imageDevice); 
    hipDeviceSynchronize();
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    hipMemcpy(image, imageDevice, imageSize, hipMemcpyDeviceToHost);
    hipFree(imageDevice);
    printf("%d %d\n", (int)WIDTH, (int)HEIGHT);
    for(int i=0; i < area; i++){
        printf("%d %d %d\n", image[i].r, image[i].g, image[i].b);
    }
    printf("Execution took %f seconds\n", cpu_time_used);
    return 0;
}
